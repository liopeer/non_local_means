
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <chrono>

__device__ float distance(float x1, float y1, float x2, float y2) {
    return sqrt((x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2));
}

__global__ void non_local_means(
    float* img, 
    float* out, 
    int height, 
    int width, 
    int radius
) {
    // coordinates of the center pixel
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // printf("x: %d, y: %d\n", x, y);

    for (int i = -radius; i <= radius; i++) {
        for (int j = -radius; j <= radius; j++) {
            int x1 = x + i;
            int y1 = y + j;
            if (x1 >= 0 && x1 < width && y1 >= 0 && y1 < height) {
                float d = distance(x, y, x1, y1);
                out[x * height + y] += img[x1 * height + y1] * exp(-d);
            }
        }
    }
}

extern "C" {
    void interface_nlm(float* img, float* out, int height, int width, int radius) {
        float* d_img;
        float* d_out;

        hipMalloc(&d_img, sizeof(float) * height * width);
        hipMalloc(&d_out, sizeof(float) * height * width);

        hipMemcpy(d_img, img, sizeof(float) * height * width, hipMemcpyHostToDevice);
        hipMemcpy(d_out, out, sizeof(float) * height * width, hipMemcpyHostToDevice);

        dim3 blockDim(16, 16);
        dim3 gridDim(ceil(width / blockDim.y), ceil(height / blockDim.x));

        non_local_means<<<gridDim, blockDim>>>(d_img, d_out, height, width, radius);
        hipDeviceSynchronize();

        hipMemcpy(out, d_out, sizeof(float) * height * width, hipMemcpyDeviceToHost);

        hipFree(d_img);
        hipFree(d_out);
    }
}